#include "hip/hip_runtime.h"
/*
 * dijkstra_cuda.cu
 *
 * Simplified CUDA implementation of Dijkstra’s algorithm.  This version
 * assumes non‑negative weights.  It uses a parallel relaxation scheme
 * similar to Bellman–Ford: all edges are processed in parallel and
 * distances are updated via atomic operations until no updates occur.
 *
 * NOTE: This is a pedagogical example.  True Dijkstra on the GPU
 * typically uses more advanced data structures (e.g. delta‑stepping or
 * bucketed priority queues) for better performance.
 */

#include "graph.h"
#include "graph_io.h"
#include "graphGen.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

/* CUDA kernel: relax all edges once.  If a shorter path is found,
 * atomically update the destination distance and set the `changed` flag. */
__global__ void relax_edges_kernel(const Edge *edges, int E, int *dist, int *changed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= E) return;
    int u = edges[idx].src;
    int v = edges[idx].dest;
    int w = edges[idx].weight;
    int du = dist[u];
    if (du != INT_MAX && du + w < dist[v]) {
        int newDist = du + w;
        int old = atomicMin(&dist[v], newDist);
        if (newDist < old) {
            *changed = 1;
        }
    }
}

int main(int argc, char **argv) {
    if (argc < 4) {
        fprintf(stderr, "Usage: %s <V> <min_w> <max_w> [density=0.005]\n", argv[0]);
        return 1;
    }
    int V = atoi(argv[1]);
    int min_w = atoi(argv[2]);
    int max_w = atoi(argv[3]);
    double density = (argc > 4) ? atof(argv[4]) : 0.005;

    if (V <= 0 || max_w < min_w || min_w < 0) {
        fprintf(stderr, "Invalid arguments. V > 0, min_w >= 0, max_w >= min_w\n");
        return 1;
    }

    Graph *g = get_or_create_graph(V, max_w, min_w, density);
    if (!g) return 1;

    int *h_dist = (int*)malloc(sizeof(int) * g->V);
    if (!h_dist) { perror("malloc"); free_graph(g); return 1; }
    for (int i = 0; i < g->V; i++) h_dist[i] = INT_MAX;
    h_dist[0] = 0;

    Edge *d_edges;
    int *d_dist;
    int *d_changed;
    hipMalloc((void**)&d_edges, sizeof(Edge) * g->E);
    hipMalloc((void**)&d_dist, sizeof(int) * g->V);
    hipMalloc((void**)&d_changed, sizeof(int));
    hipMemcpy(d_edges, g->edges, sizeof(Edge) * g->E, hipMemcpyHostToDevice);
    hipMemcpy(d_dist, h_dist, sizeof(int) * g->V, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (g->E + blockSize - 1) / blockSize;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    bool host_changed;
    do {
        host_changed = false;
        hipMemset(d_changed, 0, sizeof(int));
        relax_edges_kernel<<<gridSize, blockSize>>>(d_edges, g->E, d_dist, d_changed);
        hipDeviceSynchronize();
        int changedFlag;
        hipMemcpy(&changedFlag, d_changed, sizeof(int), hipMemcpyDeviceToHost);
        if (changedFlag != 0) host_changed = true;
    } while (host_changed);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("[dijkstra_cuda] time: %.6f s\n", ms / 1000.0f);

    hipMemcpy(h_dist, d_dist, sizeof(int) * g->V, hipMemcpyDeviceToHost);
    save_distance_vector("dijkstra_cuda", g->V, max_w, min_w, h_dist, g->V, false);

    free(h_dist);
    free_graph(g);
    hipFree(d_edges);
    hipFree(d_dist);
    hipFree(d_changed);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}